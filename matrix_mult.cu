
#include <hip/hip_runtime.h>
#include <iostream>
#include <hipblas.h>



void populateMatrix(float* h_matrix, int row, int col)
{
    for(int i = 0; i < row * col; i++)
    {
        h_matrix[i] = (rand()) % 10;
    }
}


void printMatrix(const float* matrix, int row, int col) {
    for (int i = 0; i < row; ++i) {
        for (int j = 0; j < col; ++j) {
            std::cout << matrix[i * col + j] << " ";
        }
        std::cout << std::endl;
    }
}


void matrixMult(float* h_matrixA, float* h_matrixB, float *h_matrixC, int m, int n, int k)
{
    float* d_matrixA;
    float* d_matrixB;
    float* d_matrixC;
    hipblasHandle_t cublasHandle;
    float alpha = 1.0f;
    float beta = 0.0f;

    hipblasCreate(&cublasHandle);
    int matrixASize = m * k * sizeof(float);
    int matrixBSize = n * k * sizeof(float);
    int matrixCSize = m * n * sizeof(float);
    hipMalloc(&d_matrixA, matrixASize);
    hipMalloc(&d_matrixB, matrixBSize);
    hipMalloc(&d_matrixC, matrixCSize);

    hipMemcpy(d_matrixA, h_matrixA, matrixASize, hipMemcpyHostToDevice);
    hipMemcpy(d_matrixB, h_matrixB, matrixBSize, hipMemcpyHostToDevice);

    hipblasSgemm(cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_N, n, m, k, &alpha, d_matrixB, n, d_matrixA, k, &beta, d_matrixC, n);
    
    hipMemcpy(h_matrixC, d_matrixC, matrixCSize, hipMemcpyDeviceToHost);

    hipFree(d_matrixA);
    hipFree(d_matrixB);
    hipFree(d_matrixC);
    hipblasDestroy(cublasHandle);
}


int main()
{
    int row = 3;
    int col = 3;
    float* matrixA = new float[row*col];
    float* matrixB = new float[row*col];
    float* matrixC = new float[row*col];


    populateMatrix(matrixA, row, col);
    populateMatrix(matrixB, row, col);
    
    std::cout<< "MATRIX A" << std::endl;
    printMatrix(matrixA, row, col);
    std::cout<< "MATRIX B" << std::endl;
    printMatrix(matrixB, row, col);

    matrixMult(matrixA, matrixB, matrixC, row, col, col);
    std::cout<< "MATRIX C" << std::endl;
    printMatrix(matrixC, row, col);
}