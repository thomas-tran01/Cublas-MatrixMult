
#include <hip/hip_runtime.h>
#include <iostream>
#include <hipblas.h>



void populateMatrix(float* h_matrix, int row, int col)
{
    for(int i = 0; i < row * col; i++)
    {
        h_matrix[i] = (rand()) % 100;
    }
}


void printMatrix(const float* matrix, int row, int col) {
    for (int i = 0; i < row; ++i) {
        for (int j = 0; j < col; ++j) {
            std::cout << matrix[i * col + j] << " ";
        }
        std::cout << std::endl;
    }
}


int matrixMult(float* h_matrixA, float* h_matrixB, float *h_matrixC, int m, int n, int k)
{
    return 0;
}


int main()
{
    int row = 3;
    int col = 3;
    float* matrixA = new float[row*col];
    float* matrixB = new float[row*col];
    float* matrixC;


    populateMatrix(matrixA, row, col);
    populateMatrix(matrixB, row, col);
    
    std::cout<< "MATRIX A" << std::endl;
    printMatrix(matrixA, row, col);
    std::cout<< "MATRIX B" << std::endl;
    printMatrix(matrixB, row, col);


}